#include "hip/hip_runtime.h"
///
/// matmultKernel00.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-01-27
/// Last Modified: 2011-02-23 DVN
///
/// Multiplies two matrices using CUDA: A x B = C
///
/// Copy this file and modify the MatMultKernel device function for
/// each of your experiments. 
///

#include "matmultKernel.h"

// Define a gpu kernel to perform matrix multiplication
// of A x B = C.
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){

  // matrix blocks
  float *Asub, *Bsub, *Csub;
  // Putting these into registers speeds access.
  int thread_row = threadIdx.y;
  int thread_col = threadIdx.x;

//this is where the thread blocks start in the thread space
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

//this is where they go into memory
  int footprint_row = block_row*2;
  int footprint_col = block_col*2;

//4 different C values
  float Cvalue0 = 0;
  float Cvalue1 = 0;
  float Cvalue2 = 0;
  float Cvalue3 = 0;



  Csub = &C.elements[C.stride * footprint_row * FOOTPRINT_SIZE + FOOTPRINT_SIZE * footprint_col];

  for (int m = 0;  m < (A.width / FOOTPRINT_SIZE); ++m){


  Asub = &A.elements[A.stride * FOOTPRINT_SIZE * footprint_row + FOOTPRINT_SIZE * m];
  Bsub = &B.elements[B.stride * FOOTPRINT_SIZE * m + FOOTPRINT_SIZE * footprint_col];




  
  __shared__ float shared_A[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
  __shared__ float shared_B[FOOTPRINT_SIZE][FOOTPRINT_SIZE];



  shared_A[thread_row][thread_col] = Asub[A.stride * thread_row + thread_col];
  shared_B[thread_row][thread_col] = Bsub[B.stride * thread_row + thread_col]; 
  
  shared_A[thread_row][thread_col+BLOCK_SIZE] = Asub[A.stride * thread_row + thread_col+BLOCK_SIZE];
  shared_B[thread_row][thread_col+BLOCK_SIZE] = Bsub[B.stride * thread_row + thread_col+BLOCK_SIZE]; 

  shared_A[thread_row+BLOCK_SIZE][thread_col] = Asub[A.stride * (thread_row + BLOCK_SIZE) + thread_col];
  shared_B[thread_row+BLOCK_SIZE][thread_col] = Bsub[B.stride * (thread_row + BLOCK_SIZE) + thread_col]; 
  
  shared_A[thread_row+BLOCK_SIZE][thread_col+BLOCK_SIZE] = Asub[A.stride * (thread_row + BLOCK_SIZE) + thread_col+BLOCK_SIZE];
  shared_B[thread_row+BLOCK_SIZE][thread_col+BLOCK_SIZE] = Bsub[B.stride * (thread_row + BLOCK_SIZE) + thread_col+BLOCK_SIZE]; 
   //make sure all threads 
   __syncthreads();



#pragma unroll
    for( int e = 0; e<FOOTPRINT_SIZE; e++){
//top left and top right
	Cvalue0 += shared_A[thread_row][e]*shared_B[e][e];
	Cvalue1 += shared_A[thread_row][e]*shared_B[e][thread_col+BLOCK_SIZE];
	Cvalue2 += shared_A[thread_row+BLOCK_SIZE][e]*shared_B[e][thread_col];
	Cvalue3 += shared_A[thread_row+BLOCK_SIZE][e]*shared_B[e][thread_col+BLOCK_SIZE];

    }

    // Synchronize to ensure all Cvalues have been incremented
    // before reading in the next shared_A AND shared_B BLOCKS
    __syncthreads();




  }
  // Write Csub to GLOBAL memory.
  // Each thread writes its own cell value.
 
  Csub[thread_row * C.stride + thread_col  ] = Cvalue0;
  Csub[thread_row * C.stride + thread_col  + BLOCK_SIZE] = Cvalue1;
  Csub[thread_row * C.stride + thread_col  + BLOCK_SIZE*C.stride] = Cvalue2;
  Csub[thread_row * C.stride + thread_col  + BLOCK_SIZE*C.stride + BLOCK_SIZE] = Cvalue3;

}

