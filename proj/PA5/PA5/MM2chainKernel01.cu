#include "hip/hip_runtime.h"
#include <stdio.h>
#include "MM2chainKernel.h"




__global__ void multiply(const float* A, float* partial_mult ){

int block = blockIdx.x;
int thread = threadIdx.x;


/*
if(thread == 1){
for( int i = 0; i<240; i++){
printf("%f\n", *(A+i));
}
}*/

/*
if(block==32 && thread == 1) {
for( int i = 0; i<240; i++){
printf("%f\n",  *(A+i+31*240+((BLOCK_SIZE)*(blockIdx.x))));
}
}*/

__shared__ float shared_A[B*C*4];


const float* AStart = A + ((B)*(blockIdx.x)*C*4);
//if(threadIdx.x == 1) printf("A %p Astart %p  \n ", A, AStart);

for( int i = thread; i<C*4*B; i += B){
shared_A[i] = AStart[i];
//if(block==32) printf("%f \n", shared_A[i+240*threadIdx.x]);
}



__syncthreads();

/*
if(block==32 && thread == 1) {
for( int i = 0; i<240; i++){
printf("%f %f\n", shared_A[i + 240*31], *(A+i+31*240+(BLOCK_SIZE)*(blockIdx.x)));
}
}*/



float current[] = { 1.0, 0.0, 0.0, 1.0} ;
float temp[4];

for(int i = 0; i<C*4; i += 4){

float mat00 = shared_A[i + C*4*threadIdx.x];
float mat01 = shared_A[i+1 + C*4*threadIdx.x];
float mat10 = shared_A[i+2 + C*4*threadIdx.x];
float mat11 = shared_A[i+3 + C*4*threadIdx.x];

//if(threadIdx.x == 2) printf("next matrix %f %f %f %f \n", mat00, mat01, mat10, mat11);
//if(threadIdx.x == 2) printf("should be   %f %f %f %f \n", *(A+i+threadIdx.x*240), *(A+i+1+threadIdx.x*240),*(A+i+2+threadIdx.x*240),*(A+i+3+threadIdx.x*240));


//if(threadIdx.x == 2) printf("this one \n %f %f \n %f %f \n", current[0], current[1], current[2], current[3]);
//if(threadIdx.x == 2) printf("times this  \n %f %f \n %f %f \n", mat00, mat01, mat10, mat11);

temp[0] = current[0] * mat00 + current[1]*mat10;
temp[1] = current[0] * mat01 + current[1]*mat11;
temp[2] = current[2] * mat00 + current[3]*mat10;
temp[3] = current[2] * mat01 + current[3]*mat11;

current[0] = temp[0];
current[1] = temp[1];
current[2] = temp[2];
current[3] = temp[3];


//if(threadIdx.x == 2) printf("equals \n %f %f \n %f %f \n", current[0], current[1], current[2], current[3]);


}

//if(block==32)printf("writing shared results \n %d %d \n %d %d \n", current[0], current[1], current[2], current[3]);

__shared__ float shared_results[B*4];

shared_results[threadIdx.x*4] = current[0];
shared_results[threadIdx.x*4+1] = current[1];
shared_results[threadIdx.x*4+2] = current[2];
shared_results[threadIdx.x*4+3] = current[3];

__syncthreads();
if(threadIdx.x == 0){

float current[] = { 1.0, 0.0, 0.0, 1.0} ;

	for(int i = 0; i<B; i += 4){

	temp[0] = current[0] * shared_results[threadIdx.x*4] + current[1] * shared_results[threadIdx.x*4 + 2];
	temp[1] = current[0] * shared_results[threadIdx.x*4+1] + current[1] * shared_results[threadIdx.x*4 + 3];
	temp[2] = current[2] * shared_results[threadIdx.x*4] + current[3] * shared_results[threadIdx.x*4 + 2];
	temp[3] = current[2] * shared_results[threadIdx.x*4+1] + current[3] * shared_results[threadIdx.x*4 + 3];
	
	current[0] = temp[0];
	current[1] = temp[1];
	current[2] = temp[2];
	current[3] = temp[3];
	

	//printf(" shared results \n %d %d \n %d %d \n", shared_results[threadIdx.x*4], shared_results[threadIdx.x*4+1], shared_results[threadIdx.x*4+2], shared_results[threadIdx.x*4+3]);

	}

//if(block==32)printf("block partial \n %f %f \n %f %f\n", current[0], current[1], current[2], current[3]);

float* partial_mult_start = partial_mult + 4 * block;
partial_mult_start[0] = current[0];
partial_mult_start[1] = current[1];
partial_mult_start[2] = current[2];
partial_mult_start[3] = current[3];



}



}
