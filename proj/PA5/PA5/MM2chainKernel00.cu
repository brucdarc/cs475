#include "hip/hip_runtime.h"
#include <stdio.h>
#include "MM2chainKernel.h"




__global__ void multiply(const float* A, float* partial_mult ){

int block = blockIdx.x;
int thread = threadIdx.x;
int BLOCK_SIZE =  gridDim.x;



__shared__ float shared_A[B*C*4];
__shared__ float shared_results[B*4];





const float* AStart = A + ((B)*(blockIdx.x)*C*4);


for( int i = 0; i<C*4; i++){
shared_A[i+thread*C*4] = AStart[i+thread*C*4];

}



__syncthreads();



float current[] = { 1.0, 0.0, 0.0, 1.0} ;
float temp[4];

for(int i = 0; i<C*4; i += 4){

float mat00 = shared_A[i + C*4*threadIdx.x];
float mat01 = shared_A[i+1 + C*4*threadIdx.x];
float mat10 = shared_A[i+2 + C*4*threadIdx.x];
float mat11 = shared_A[i+3 + C*4*threadIdx.x];


temp[0] = current[0] * mat00 + current[1]*mat10;
temp[1] = current[0] * mat01 + current[1]*mat11;
temp[2] = current[2] * mat00 + current[3]*mat10;
temp[3] = current[2] * mat01 + current[3]*mat11;

current[0] = temp[0];
current[1] = temp[1];
current[2] = temp[2];
current[3] = temp[3];


}


shared_results[threadIdx.x*4] = current[0];
shared_results[threadIdx.x*4+1] = current[1];
shared_results[threadIdx.x*4+2] = current[2];
shared_results[threadIdx.x*4+3] = current[3];

__syncthreads();
if(threadIdx.x == 0){

float current[] = { 1.0, 0.0, 0.0, 1.0} ;

	for(int i = 0; i<B; i += 4){

	temp[0] = current[0] * shared_results[threadIdx.x*4] + current[1] * shared_results[threadIdx.x*4 + 2];
	temp[1] = current[0] * shared_results[threadIdx.x*4+1] + current[1] * shared_results[threadIdx.x*4 + 3];
	temp[2] = current[2] * shared_results[threadIdx.x*4] + current[3] * shared_results[threadIdx.x*4 + 2];
	temp[3] = current[2] * shared_results[threadIdx.x*4+1] + current[3] * shared_results[threadIdx.x*4 + 3];
	
	current[0] = temp[0];
	current[1] = temp[1];
	current[2] = temp[2];
	current[3] = temp[3];
	


	}


float* partial_mult_start = partial_mult + 4 * block;
partial_mult_start[0] = current[0];
partial_mult_start[1] = current[1];
partial_mult_start[2] = current[2];
partial_mult_start[3] = current[3];



}



}
